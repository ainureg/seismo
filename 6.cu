
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <complex.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <algorithm>
#define h_x 0.01
#define h_y 0.005
#define N 100 //размер расчетной плоскости по X
#define M 400 //размер расчетной плоскости по Y
#define tmax 5 // нужный момент времени
double c(double x, double y);
double v(double t, double x);
void save(const char *prefix, int step, double *a);
int ind (int i, int j) {
return j*(N+1) + i;
}
int main(int argc, char **argv){//int main( int argc, char * argv [] ){
int i;
double t=h_x/2;
double *a = new double[(N+1) * (M+1)];
// U = (double**)malloc((N) * sizeof(double*));
/* // for (i=0;i<N;i++){
U[i]=(double*)malloc((M) * sizeof(double));
}
*/
std::cout << "Bang!" << std::endl;
for (int step = 0; step < 1000; step++){
for (int i = 0; i < N+1; i++)
for (int j = 0; j < M+1; j++){
double x = h_x*i;
double y = h_y*j;
a[ind(i,j)] = x + y + step / 1000.;
}
if (step == (step / 10) * 10 )
save("output", step, a);
}
}
//---------------------------------------------------------------------------//
template<typename T>
static void put(std::fstream &f, const T value) {
union {
char buf[sizeof(T)];
T val;
} helper;
helper.val = value;
std::reverse(helper.buf, helper.buf + sizeof(T));
f.write(helper.buf, sizeof(T));
}
void save(const char *prefix, int step, double *a){
char buffer[50];
sprintf(buffer, "%s.%05d.vtk", prefix, step);
std::fstream f(buffer, std::ios::out);
if (!f){
std::cerr << "Unable to open file " << buffer << std::endl;
return;
}
f << "# vtk DataFile Version 3.0" << std::endl;
f << "U data" << std::endl;
f << "BINARY" << std::endl;
f << "DATASET STRUCTURED_POINTS" << std::endl;
f << "DIMENSIONS " << N+1 << " " << M+1 << " 1" << std::endl;
f << "SPACING " << h_x << " " << h_y << " 1" << std::endl;
f << "ORIGIN 0 0 0" << std::endl;
f << "POINT_DATA " << (N+1) * (M+1) << std::endl;
f << "SCALARS u double" << std::endl;
f << "LOOKUP_TABLE default" << std::endl;
for (int j = 0 ; j < M+1; j++){
for (int i = 0; i < N+1; i++)
put(f, a[ind(i,j)]);
}
}
double v(double t, double x) {
if (5*t<2*M_PI){
return sin(5*t)*exp(-30*(x-0.5)*(x-0.5));
}
else return 0.0;
}
double c(double x, double y) {
if (1.0<y<=1.2){
return 0.8;
}
else if( (0.5<y<=0.8)&(0.2<x<=0.5) ){
return 1.0;
}
else return 0.5;
} 
