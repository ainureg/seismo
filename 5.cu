
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <complex.h>
#include <math.h>
#define N 400 //размер расчетной прямой
#define tmax 10 // нужный момент времени

float mu=100;
float const w = 5;
float h=(float) 2/N;//шаг по прямой;
void KMatr(int n, hipFloatComplex** M);
hipFloatComplex prm (float x);
hipFloatComplex sigm(float x);
float gf(float m);
float cf(float x);
void MMatr(int n, hipFloatComplex** M);
void FMatr(int n, hipFloatComplex* M);
int solve(int n, hipFloatComplex *top, hipFloatComplex *mid, hipFloatComplex *bot, hipFloatComplex *b, hipFloatComplex *x);

int main( int argc, char *  argv []  ){
    
    
    int k,i,n,j;
    k=(int)pow(2,(int)ceil(log(N-1)/log(2)));
    n=k+1;
    hipFloatComplex **K, **M, *F, **A;
    K = (hipFloatComplex**)malloc((n) * sizeof(hipFloatComplex*));
    M = (hipFloatComplex**)malloc((n) * sizeof(hipFloatComplex*));
    F = (hipFloatComplex*) malloc((n) * sizeof(hipFloatComplex)) ;
    A = (hipFloatComplex**)malloc((n) * sizeof(hipFloatComplex*));
    
    for (i=0;i<n;i++){
        K[i]=(hipFloatComplex*)malloc((n) * sizeof(hipFloatComplex));
        M[i]=(hipFloatComplex*)malloc((n) * sizeof(hipFloatComplex));
        A[i]=(hipFloatComplex*)malloc((n) * sizeof(hipFloatComplex));
    }
    
     KMatr(N, K);
     MMatr(N, M);
     FMatr(N, F);
     for (i=0;i<N;i++){
        for (j=0;j<N;j++){
            K[i][j]=hipCdivf(K[i][j],make_hipFloatComplex(h,0)) ;
            M[i][j]=hipCmulf(M[i][j],make_hipFloatComplex(h/6,0));
            } 
        }
     for (i=0;i<N;i++){
       for (j=0;j<N;j++){
           A[i][j]=hipCaddf( hipCmulf( make_hipFloatComplex( -w*w,0), M[i][j]), K[i][j]);
        }        
     }
     hipFloatComplex *a,*b,*c,*u;
    a = (hipFloatComplex*)malloc((n)  * sizeof(hipFloatComplex));
    b = (hipFloatComplex*)malloc((n)  * sizeof(hipFloatComplex));
    c = (hipFloatComplex*)malloc((n)  * sizeof(hipFloatComplex));
    u = (hipFloatComplex*)malloc((n)  * sizeof(hipFloatComplex));
    
     for (i=1;i<(N-1);i++){
        a[i+1]=A[i+1][i];
        b[i]=A[i][i];
        c[i-1]=A[i][i-1];
    }
    
    a[0]=make_hipFloatComplex(0,0);
    a[1]=A[1][0];
    b[0]=A[0][0];
    b[N-1]=A[N-1][N-1];
    c[N-2]=A[N-1][N-2];
    c[N-1]=make_hipFloatComplex(0,0);
     
    for (j=N;j<(n);j++) {
            a[j]=make_hipFloatComplex(0,0);
            b[j]=make_hipFloatComplex(1,0);
            c[j]=make_hipFloatComplex(0,0);
            F[j]=make_hipFloatComplex(0,0);
      }
      
      
    int l;
    l=solve (n,a,b,c,F,u);
}     

int solve(int n, hipFloatComplex *top, hipFloatComplex *mid, hipFloatComplex *bot, hipFloatComplex *b, hipFloatComplex *x){
    return 0;
}

void MMatr(int n, hipFloatComplex** M){
   int i;
   hipFloatComplex t1,t2;
   for (i=0;i<(n-1);i++) {
        t1=hipCmulf(make_hipFloatComplex( gf(i+1.5),0), prm(i+1.5));
        t2=hipCmulf( make_hipFloatComplex(gf(i+0.5),0) , prm(i+0.5));
        
       
        M[i][i]=hipCmulf(make_hipFloatComplex(2,0), hipCaddf( t1 ,t2 ));
        M[i+1][i]=t1;
        M[i][i+1]=t1;
        }
    M[n-1][n-1]=hipCmulf(make_hipFloatComplex( 2*gf(n-0.5),0), prm(n-0.5)) ;
    }
hipFloatComplex prm ( float x){
    return hipCdivf(hipCaddf( make_hipFloatComplex(w,0), hipCmulf(make_hipFloatComplex(0,-1), sigm(x*h) )) , make_hipFloatComplex(w,0));
    }        
hipFloatComplex sigm(float x){
if (x <= 1)
    return  make_hipFloatComplex (0,0);
else
    return make_hipFloatComplex( mu*(x-1)*(x-1),0);
    }    
float gf(float m){
    return ( ( 1/cf(m*h) ) / cf(m*h) );
    }       
float cf(float x){
    if (x<=1)
    return (0.1 +3.6*(x-0.5)*(x-0.5));
    else
    return 1;
     }    
void KMatr(int n, hipFloatComplex ** M){
    int i;
    hipFloatComplex t1,t2;

    for (i=0;i<(n-1);i++){
        t1=hipCdivf(make_hipFloatComplex(1,0), prm(i+0.5));
        t2=hipCdivf(make_hipFloatComplex(1,0), prm(i+1.5));
        
        
        M[i][i]= hipCaddf(t1,t2);
        M[i+1][i]=hipCdivf(make_hipFloatComplex(-1,0), prm(i+1.5));
        M[i][i+1]=hipCdivf(make_hipFloatComplex(-1,0), prm(i+1.5));
}
    M[n-1][n-1]=hipCdivf(make_hipFloatComplex(1,0), prm(n-0.5));
    }     
void FMatr(int n, hipFloatComplex * M){
    M[0]=hipCaddf( hipCmulf(make_hipFloatComplex(w*w*h*gf(0.5)/6,0) , prm(0.5) ) , hipCmulf(make_hipFloatComplex(1/h,0),prm(0.5)));
    }    